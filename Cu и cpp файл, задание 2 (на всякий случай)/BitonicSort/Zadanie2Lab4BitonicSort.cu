#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <iomanip>
#include <ctime>
#include <cstdlib>

// ������ ��� �������� ������ CUDA.
#define CUDA_CHECK(call) \
         do { \
             hipError_t err = call; \
             if (err != hipSuccess) { \
                 std::cerr << "CUDA error in " << #call << ": " \
                           << hipGetErrorString(err) << std::endl; \
                 exit(EXIT_FAILURE); \
             } \
         } while (0)

     // ���� CUDA ��� ���� Bitonic Sort.
__global__ void bitonicSortStep(int* arr, int n, int k, int j) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int ixj = idx ^ j;
    if (ixj > idx && idx < n && ixj < n) {
        bool up = ((idx & k) == 0);
        int a = arr[idx];
        int b = arr[ixj];
        if ((a > b) == up) {
            arr[idx] = b;
            arr[ixj] = a;
        }
    }
}

int main(int argc, char* argv[]) {
    // ����������� ���������.
    std::cerr << "Program started on host: " << std::getenv("HOSTNAME") << "\n";
    std::cerr << "CUDA_VISIBLE_DEVICES=" << (std::getenv("CUDA_VISIBLE_DEVICES") ? std::getenv("CUDA_VISIBLE_DEVICES") : "not set") << "\n";
    std::cerr << "LD_LIBRARY_PATH=" << (std::getenv("LD_LIBRARY_PATH") ? std::getenv("LD_LIBRARY_PATH") : "not set") << "\n";

    // ��������� ������� ���������.
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <array_size>\n"
            << "Example: " << argv[0] << " 262144\n";
        return 1;
    }
    int N = std::atoi(argv[1]);
    if (N <= 100000) {
        std::cerr << "Error: array_size must be greater than 100000\n";
        return 1;
    }
    // Bitonic Sort �������, ����� ������ ��� �������� ������.
    int pow2N = 1;
    while (pow2N < N) pow2N <<= 1;
    if (pow2N != N) {
        std::cerr << "Error: array_size must be a power of 2 for Bitonic Sort, got " << N << "\n";
        return 1;
    }
    std::cerr << "Starting with array_size=" << N << "\n";

    // ��������� ����������� GPU.
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipGetDeviceCount: " << hipGetErrorString(err) << "\n";
        return 1;
    }
    if (deviceCount == 0) {
        std::cerr << "Error: No CUDA-capable devices found\n";
        return 1;
    }
    std::cerr << "Found " << deviceCount << " CUDA device(s)\n";
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, i));
        std::cerr << "Device " << i << ": " << prop.name
            << ", Compute Capability: " << prop.major << "." << prop.minor
            << ", Memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB\n";
    }
    CUDA_CHECK(hipSetDevice(0));
    std::cerr << "Selected device 0\n";

    // ������������� ���������� ��������� �����.
    std::srand(static_cast<unsigned>(std::time(nullptr)));

    // ��������� �������� �������.
    std::vector<int> h_array(N);
    for (int i = 0; i < N; ++i) {
        h_array[i] = rand() % 1000;
    }

    // ��������� ����� ��� �������� ������������.
    int sum_orig = 0;
    for (int v : h_array) sum_orig += v;

    // ������������ ���������� (Bitonic Sort).
    std::cerr << "Running parallel Bitonic Sort...\n";
    int* d_array;
    CUDA_CHECK(hipMalloc(&d_array, N * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_array, h_array.data(), N * sizeof(int), hipMemcpyHostToDevice));

    const int threadsPerBlock = 512;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    double par_time = 0.0;
    int sum_par = 0;
    for (int run = 0; run < 100; ++run) {
        CUDA_CHECK(hipMemcpy(d_array, h_array.data(), N * sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipEventRecord(start));
        for (int k = 2; k <= N; k <<= 1) {
            for (int j = k >> 1; j > 0; j >>= 1) {
                bitonicSortStep << <blocks, threadsPerBlock >> > (d_array, N, k, j);
                CUDA_CHECK(hipGetLastError());
                CUDA_CHECK(hipDeviceSynchronize());
            }
        }
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        float ms = 0;
        CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
        par_time += ms * 1e-3f;
        if (run == 0) {
            std::vector<int> h_result(N);
            CUDA_CHECK(hipMemcpy(h_result.data(), d_array, N * sizeof(int), hipMemcpyDeviceToHost));
            for (int v : h_result) sum_par += v;
        }
    }
    par_time /= 100.0;

    // ����� �����������.
    std::cout << "\n=== Parallel Bitonic Sort (average over 100 runs) ===\n";
    std::cout << std::setw(12) << "Array size"
        << std::setw(12) << "Sum"
        << std::setw(20) << "Avg time (s)\n"
        << std::string(44, '-') << "\n";
    std::cout << std::setw(12) << N
        << std::setw(12) << sum_par
        << std::setw(20) << std::fixed << std::setprecision(6) << par_time
        << "\n\n";

    // �������� ������������.
    if (sum_orig != sum_par) {
        std::cerr << "Error: Sum mismatch! Original: " << sum_orig
            << ", Parallel: " << sum_par << "\n";
        return 1;
    }

    CUDA_CHECK(hipFree(d_array));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    std::cerr << "Program completed successfully\n";
    return 0;
}